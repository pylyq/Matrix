#include "hip/hip_runtime.h"
// Copyright 2018 The Matrix Authors
// This file is part of the Matrix library.
//
// The Matrix library is free software: you can redistribute it and/or modify
// it under the terms of the GNU Lesser General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// The Matrix library is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU Lesser General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public License
// along with the Matrix library. If not, see <http://www.gnu.org/licenses/>.
#include<stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include "pub.h"
#include "min.h"

#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/tuple.h>
#include <thrust/fill.h>

__global__ void kernel_ComputeMin(float* gpu_in, int D1, float* gpu_out, int D2){
    
    int s = threadIdx.x;
    int index_out = blockIdx.x * blockDim.x + s;
    
    __shared__ float sdata[1024];

    if(index_out < D1){
        sdata[s] = gpu_in[index_out];
    }

    int stop = blockDim.x;
    if(blockDim.x * (blockIdx.x + 1) > D1)
        stop = D1 - (blockDim.x * blockIdx.x);

    __syncthreads();
    unsigned int s0 = blockDim.x;
    while(s0 >= stop){
        s0 = (s0>>1);
    }
    
    __syncthreads();

    if(s < (stop-s0) && sdata[s] > sdata[s+s0])
        sdata[s] = sdata[s+s0];

    __syncthreads();

    for(s0 = (s0>>1); s0>0; s0>>=1){
        if(s < s0 && sdata[s] > sdata[s+s0])
            sdata[s] = sdata[s+s0];
        __syncthreads();
    }
    if(s == 0){
        gpu_out[blockIdx.x] = sdata[0];
    }
}


__global__ void kernel_ComputeArgMin(float* gpu_in, int* gpu_idxin, int D1, float* gpu_out, int* gpu_idxout, int D2){
    
    int s = threadIdx.x;
    int index_out = blockIdx.x * blockDim.x + s;
    
    __shared__ float sdata[1024];
    __shared__ unsigned int sidx[1024];

    if(index_out < D1){
        sdata[s] = gpu_in[index_out];
        sidx[s] = gpu_idxin[index_out];
    }

    int stop = (blockIdx.x+1==gridDim.x)?(D1-(blockDim.x* blockIdx.x)):blockDim.x;

    __syncthreads();

    unsigned int s0 = blockDim.x;
    while(s0 >= stop){
        s0 = (s0>>1);
    }
    
    __syncthreads();

    if(s < (stop-s0) && sdata[s] > sdata[s+s0]){
        sdata[s] = sdata[s+s0];
        sidx[s] = sidx[s+s0];
    }

    __syncthreads();
    for(s0 = (s0>>1); s0>0; s0>>=1){
        if(s < s0 && sdata[s] > sdata[s+s0]){
            sdata[s] = sdata[s+s0];
            sidx[s] = sidx[s+s0];
        }
        __syncthreads();
    }
    if(s == 0){
        gpu_out[blockIdx.x] = sdata[0];
        gpu_idxout[blockIdx.x] = sidx[0];
    }
}

__global__ void kernel_GiveIndex(int* vectorindex, int Dn){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < Dn){
        vectorindex[idx] = idx;
    }
}

__host__ void host_ComputeArgMin(float* gpu_left, int* gpu_leftidx, int D1, float* gpu_right, int* gpu_rightidx, thrust::tuple<float,int>* p1){
    
    int totals = D1;
    float* temp;
    float* left;
    float* right;
    int* tempidx;
    int* leftidx;
    int* rightidx;
    left = gpu_left;
    right = gpu_right;
    leftidx = gpu_leftidx;
    rightidx = gpu_rightidx;
    kernel_GiveIndex<<<((totals-1)>>10)+1,1024>>>(leftidx,D1);
    while(totals > 1){
        int nexttotals = ((totals-1)>>10) + 1;
        kernel_ComputeArgMin<<<nexttotals,1024>>>(left,leftidx,totals,right,rightidx,nexttotals);
        temp = left;
        left = right;
        right = temp;

        tempidx = leftidx;
        leftidx = rightidx;
        rightidx = tempidx;
        totals = nexttotals;
    }
    
    int index_out;
    float ifgain;
    CheckCudaError(hipMemcpy(&ifgain, gpu_left, sizeof(float), hipMemcpyDeviceToHost), "copy from gpu_min to debug failed\n");
    CheckCudaError(hipMemcpy(&index_out, gpu_leftidx, sizeof(int), hipMemcpyDeviceToHost), "copy from gpu_minvalue to debugid failed\n");
    (*p1) = thrust::make_tuple(ifgain,index_out);
    //printf("gpu calc : ifgain = %lf, idx = %d\n",ifgain,index_out);
}
