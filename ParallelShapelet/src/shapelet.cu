#include "hip/hip_runtime.h"
// Copyright 2018 The Matrix Authors
// This file is part of the Matrix library.
//
// The Matrix library is free software: you can redistribute it and/or modify
// it under the terms of the GNU Lesser General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// The Matrix library is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU Lesser General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public License
// along with the Matrix library. If not, see <http://www.gnu.org/licenses/>.
#include<stdio.h>
#include "shapelet.h"
#include "pub.h"
#include "prunedtw.h"
#include "infogain.h"
#include "euclid.h"
#include "min.h"
#include "distanceoncpu.h"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <string>

#define ShareMemThrehold 48*1024

#define INFOGAINBLOCKSIZE 32

//#define BankConflictStride 17

const int coalescefor128B = 128/sizeof(float);

#define ENTIRE_STARTTIME hipEvent_t ENTIRE_Mstart0, ENTIRE_Mstop0; \
                      hipEventCreate(&ENTIRE_Mstart0); \
                  hipEventCreate(&ENTIRE_Mstop0); \
                  hipEventRecord(ENTIRE_Mstart0, 0);

#define ENTIRE_STOPTIME hipEventRecord(ENTIRE_Mstop0, 0); \
                     hipEventSynchronize(ENTIRE_Mstop0); \
                 float ENTIRE_Mtime0; \
                 hipEventElapsedTime(&ENTIRE_Mtime0, ENTIRE_Mstart0, ENTIRE_Mstop0); \
                 std::cout << "output:$$$$$$,The entire elapsed time: " << ENTIRE_Mtime0 << "ms.(including variable create and distroy)" << std::endl; \
                 hipEventDestroy(ENTIRE_Mstart0); \
                 hipEventDestroy(ENTIRE_Mstop0);

#define STARTTIME hipEvent_t Mstart1, Mstop1; \
                      hipEventCreate(&Mstart1); \
                  hipEventCreate(&Mstop1); \
                  hipEventRecord(Mstart1, 0);

#define STOPTIME hipEventRecord(Mstop1, 0); \
                     hipEventSynchronize(Mstop1); \
                 float Mtime1; \
                 hipEventElapsedTime(&Mtime1, Mstart1, Mstop1); \
                 std::cout << "The entire calc time: " << Mtime1 << "ms.(no create and distroy)" << std::endl; \
                 hipEventDestroy(Mstart1); \
                 hipEventDestroy(Mstop1);

#define TEMPSTARTTIME hipEventRecord(temp_Mstart, 0);

#define TEMPSTOPTIME(x)  hipEventRecord(temp_Mstop, 0); \
                     hipEventSynchronize(temp_Mstop); \
                    hipEventElapsedTime(&temp_Mtime, temp_Mstart, temp_Mstop); \
                     std::cout << (x) << ": "<< temp_Mtime << "ms." << std::endl;



void Batch_CalcMinInfogain(float* gpu_data, char* gpu_label,float* gpu_cmpMinDist, float* gpu_infogain, float* gpu_dividepoint, int* gpu_leftis,int* gpu_glbidx, float* gpu_tempmin, int* gpu_tempminidx, int w, int N, int L, int lo, int hi, int shapelet_num, float outthrehold, float* lastinfogain, TPOUT* p3, int suma, int sumb, hiprandState_t* states, int idxst, int idxend, std::string func, int* gpu_arrindex, int* arrindex, float* time1, float* time2, float* gpu_fortranspose, int CoalescedForL_lo_1){

    hipEvent_t temp_Mstart, temp_Mstop;
    float temp_Mtime;
    hipEventCreate(&temp_Mstart);
    hipEventCreate(&temp_Mstop);
    
    fillarrwith<<<(shapelet_num-1)/1024+1,1024>>>(gpu_infogain,shapelet_num,INFINITY);
    CheckCudaError(hipGetLastError(),"After Fill INFINITY.\n");
    CheckCudaError(hipDeviceSynchronize(),"After Fill INFINITY.\n");
   
    TEMPSTARTTIME;

    int BlockDimForX;
    int coalesceforN = ((N-1)/coalescefor128B+1)*coalescefor128B;    
    int size;
    BlockDimForX = NumsOfThreadPerBlock(L-lo+1);
    if(func == "euclid"){
        dim3 dimgrid(idxend - idxst,hi-lo);
        dim3 dimblock(BlockDimForX,1);
        size = 2 * L * sizeof(float) + L * sizeof(int);
        printf("%s need %d bytes share mem, dimgrid = (%d,%d),dimblock = (%d,%d)\n",func.c_str(),size,dimgrid.x,dimgrid.y,dimblock.x,dimblock.y);
        kernel_ComputeDist<<<dimgrid,dimblock,size>>>(gpu_data,gpu_infogain,gpu_dividepoint,gpu_leftis,N,L,lo,hi,outthrehold,gpu_cmpMinDist,idxst,gpu_arrindex);
        CheckCudaError(hipGetLastError(),"1.After ComputeDist.\n");
        CheckCudaError(hipDeviceSynchronize(),"1.After ComputeDist.\n");
        goto CALCINFOGAIN;
    }
   
    if(func == "euclidtranspose"){
        
        dim3 dimgrid(idxend - idxst,hi-lo);
        dim3 dimblock(CoalescedForL_lo_1,1);
        size = 3 * L * sizeof(float);
        /*
        if(size > ShareMemThrehold){
            printf("euclidtranspose share memory can not afford\n");
        }
        */
        printf("%s need %d bytes share mem, dimgrid = (%d,%d),dimblock = (%d,%d)\n",func.c_str(),size,dimgrid.x,dimgrid.y,dimblock.x,dimblock.y);
        kernel_ComputeDistSaveBlk<<<dimgrid,dimblock,size>>>(gpu_data, N,L, lo, hi,  gpu_fortranspose, idxst, gpu_arrindex,CoalescedForL_lo_1);

        CheckCudaError(hipGetLastError(),"1.After ComputeDist.\n");
        CheckCudaError(hipDeviceSynchronize(),"1.After ComputeDist.\n");
        size = L*sizeof(int) + (L-lo+1)*33*sizeof(float);
        dim3 dimblocktrans(32, CoalescedForL_lo_1/32);
        kernel_Transpose<<<dimgrid,dimblocktrans,size>>>(gpu_fortranspose, gpu_cmpMinDist, N, L, lo, hi, idxst, gpu_arrindex, coalesceforN,CoalescedForL_lo_1);

        //kernel_ComputeDist_AddTranspose<<<dimgrid,dimblock,size>>>(gpu_data,gpu_infogain,gpu_dividepoint,gpu_leftis,N,L,lo,hi,outthrehold,gpu_cmpMinDist,idxst,gpu_arrindex);
        CheckCudaError(hipGetLastError(),"1.After transpose.\n");
        CheckCudaError(hipDeviceSynchronize(),"1.After transpose.\n");
        goto CALCINFOGAIN;
    }

   /* 
    size = N*2*(w+2)*sizeof(float) + L* sizeof(int);
    BlockDimForX = NumsOfThreadPerBlock(N); 
//  if(size < ShareMemThrehold && N <= 1024){
    if(func == "dtw" && BlockDimForX <= 1024){
        dim3 dimgrid(idxend - idxst,L-lo+1);
        dim3 dimblock(BlockDimForX,1);
        printf("%s need %d bytes share mem, dimgrid = (%d,%d),dimblock = (%d,%d)\n",func.c_str(),size,dimgrid.x,dimgrid.y,dimblock.x,dimblock.y);
        kernel_Computedtwsperblock<<<dimgrid,dimblock,size>>>(gpu_data, gpu_data, w, N, L, lo , hi,gpu_cmpMinDist, idxst, gpu_arrindex);
        CheckCudaError(hipGetLastError(),"2.After compute dtw block.\n");
        CheckCudaError(hipDeviceSynchronize(),"2.After compute dtw block.\n");
        goto CALCINFOGAIN;
    }
   */ 
    BlockDimForX = 64;
//    int stride = 2*(w+2) + 1;
    size = BlockDimForX* (2*w+5) *sizeof(float) + L * sizeof(int);
//  if(size < ShareMemThrehold){
    if(func == "geneticdtw"){
        dim3 dimgrid(idxend - idxst, L-lo+1);
        dim3 dimblock(BlockDimForX,1);
        printf("%s need %d bytes share mem, dimgrid = (%d,%d),dimblock = (%d,%d)\n",func.c_str(),size,dimgrid.x,dimgrid.y,dimblock.x,dimblock.y);
        kernel_ComputedtwsperblockforAll<<<dimgrid,dimblock,size>>>(gpu_data, gpu_data, w, N, L, lo ,hi, gpu_cmpMinDist,idxst, gpu_arrindex);
        //kernel_ComputedtwsperblockforlargerW<<<dimgrid,dimblock,size>>>(gpu_data, gpu_data, w, N, L, lo ,hi, gpu_cmpMinDist,idxst, gpu_arrindex);
        CheckCudaError(hipGetLastError(),"3.After compute dtw block.\n");
        CheckCudaError(hipDeviceSynchronize(),"3.After compute dtw block.\n");
    }else{
        printf("The data is too large to provide a suitable method.\n");
        return;
    }

CALCINFOGAIN:
    TEMPSTOPTIME("CALC DISTANCE");
    (*time1) += temp_Mtime;
    if(N <= 40){
        //CALC INFOGAIN
        dim3 dimgrid1((shapelet_num-1)/INFOGAINBLOCKSIZE+1,1);
        dim3 dimblock1(1,INFOGAINBLOCKSIZE);
        //printf("dim (%d,%d),(%d,%d)\n",dimgrid1.x,dimgrid1.y,dimblock1.x,dimblock1.y); 
        int size2 = INFOGAINBLOCKSIZE * N + L*sizeof(int);
        TEMPSTARTTIME;
        printf("calc infogain need %d bytes, when N = %d and blockDim.y = %d\n",size2,N,INFOGAINBLOCKSIZE); 
        kernel_Computeifgain<<<dimgrid1, dimblock1, size2>>>(gpu_cmpMinDist,gpu_label, N, shapelet_num, gpu_infogain,gpu_dividepoint,gpu_leftis,gpu_glbidx,outthrehold,L,lo,hi,size2, coalesceforN,suma,sumb,states,w,idxst,gpu_arrindex);
        CheckCudaError(hipGetLastError(),"After compute ifgain.\n");
        CheckCudaError(hipDeviceSynchronize(),"After compute ifgain.\n");
        /*
        float bbbbbb[coalesceforN];
        CheckCudaError(hipMemcpy(bbbbbb,gpu_cmpMinDist+(coalesceforN*63571),coalesceforN*sizeof(float),hipMemcpyDeviceToHost),"aaaaa\n");
        for(int i = 0;i < coalesceforN;i++){
            printf("bbbbbb:%lf\n",bbbbbb[i]);
        }
        */
        TEMPSTOPTIME("CALC INFOGAIN");
        (*time2) += temp_Mtime;
        //CALC MIN  
        thrust::tuple<float,int> p1;
        host_ComputeArgMin(gpu_infogain,gpu_glbidx,shapelet_num,gpu_tempmin,gpu_tempminidx,&p1);
        CheckCudaError(hipGetLastError(),"After ArgMin\n");
        CheckCudaError(hipDeviceSynchronize(),"After ArgMin\n");

        printf("this infogain = %lf\n",thrust::get<0>(p1));
        //COMPARE WITH LAST BATCH
        if(thrust::get<0>(p1) < (*lastinfogain)){
            (*lastinfogain) = thrust::get<0>(p1);
            float dividepoint;
            int leftis;
            int index_out = thrust::get<1>(p1);

            CheckCudaError(hipMemcpy(&dividepoint, gpu_dividepoint+index_out, sizeof(float), hipMemcpyDeviceToHost), "copy from gpu_dividepoint to dividepoint failed\n");
            CheckCudaError(hipMemcpy(&leftis, gpu_leftis+index_out, sizeof(int), hipMemcpyDeviceToHost), "copy from gpu_leftis to leftis failed\n");
            
            //thrust::tuple<int,int,int> p2 = index_decompose(index_out,N,L-lo+1,hi-lo);
            thrust::tuple<int,int,int> p2 = index_decomposeNoloss(index_out,arrindex,L-lo+1);
            (*p3) = thrust::make_tuple(thrust::get<0>(p2)+idxst,thrust::get<2>(p2)+lo,thrust::get<1>(p2),thrust::get<0>(p1),dividepoint,leftis);
        }
    }else{
        
        int blocksize = 100;
        float* cpu_cmpMinDist = (float*)malloc(blocksize * coalesceforN * sizeof(float));
        char* cpu_label = (char*)malloc(blocksize * N * sizeof(char));
   
         
        for(int offset = 0; offset < shapelet_num; offset += blocksize){
            
            CheckCudaError(hipMemcpy(cpu_cmpMinDist, gpu_cmpMinDist + (offset * coalesceforN) , blocksize * coalesceforN * sizeof(float), hipMemcpyDeviceToHost), "cpu_cmpMinDist copy\n");
            CheckCudaError(hipMemcpy(cpu_label, gpu_label, blocksize * N * sizeof(char), hipMemcpyDeviceToHost),"cpu_label copy\n");
            
            T3ddi p1;
            for(int i = 0; i < blocksize; i++){
                thrust::tuple<int,int,int> n123 = index_decomposeNoloss(offset + i, arrindex, L-lo+1);
                int idx = thrust::get<0>(n123) + idxst;
                int st = thrust::get<1>(n123);
                int len = thrust::get<2>(n123) + lo;
                
                float* dist = cpu_cmpMinDist + i * coalesceforN;
                char* label = cpu_label + i * N;
                thrust::sort_by_key(thrust::seq, dist, dist + N, label);
                BestSplitInfogain(dist, N, label, &p1, suma, sumb);
                if(thrust::get<0>(p1) < outthrehold){
                    printf("infogain = %lf,dividepoint = %lf, leftis = %d, idx = %d, st = %d, len = %d, index_out = %d,(%d,%d,%d,%d)\n",thrust::get<0>(p1), thrust::get<1>(p1),thrust::get<2>(p1),idx,st,len,offset+i,lo,hi,N,L);
                }
                if(thrust::get<0>(p1) < (*lastinfogain)){
                    (*lastinfogain) = thrust::get<0>(p1);
                    (*p3) = thrust::make_tuple(idx,len,st,thrust::get<0>(p1),thrust::get<1>(p1),thrust::get<2>(p1));
                }  
            }            

        }
        
    
    }

    hipEventDestroy(temp_Mstart);
    hipEventDestroy(temp_Mstop);

}

void PreCalcIndex(int lo, int hi, int L, int* arr){
    arr[0] = 0;
    //printf("%d\n",arr[0]);
    for(int i = 0;i < (L-lo+1);i++){
        arr[i+1] = min(hi-lo,L-lo+1-i) + arr[i];
    //    printf("%d\n",arr[i+1]);
    }
}


void apply(float* data, char* label, int w, int N, int L, int lo, int hi, int shapelet_num, float outthrehold, TPOUT* p3, std::string func){

    //move from gpu to cpu for count a,b
    int suma = 0;
    int sumb = 0;
    for(int i = 0;i < N;i++){
        if(label[i] == 0)
            suma ++;
        else
            sumb ++;
    }

    if(func == "cpu"){
        printf("cpu calc#########################\n");
        host_calcdtws(data,data,label,N,L,lo,hi,w,suma,sumb,p3);
        return;
    }

    ENTIRE_STARTTIME;

    int* arrindex = (int*)malloc(L*sizeof(int));
    PreCalcIndex(lo,hi,L,arrindex);
    int idxstep = shapelet_num/arrindex[L-lo+1];
    
    int* gpu_arrindex;
    CheckCudaError(hipMalloc((void **)&gpu_arrindex, L*sizeof(int)),"cuda malloc gpu_arrindex failed\n");
    CheckCudaError(hipMemcpy(gpu_arrindex, arrindex, L*sizeof(int), hipMemcpyHostToDevice), "copy from data to gpu_data failed\n");
    
    float* gpu_data = NULL;
    char* gpu_label = NULL;
        
    int coalesceforN = ((N-1)/coalescefor128B+1)*coalescefor128B;    
    //int shapelet_num = N*(L-lo+1)*(hi-lo);

    CheckCudaError(hipMalloc((void **)&gpu_data,N*L*sizeof(float)),"cuda malloc gpu_data failed\n");
    CheckCudaError(hipMemcpy(gpu_data, data, N*L*sizeof(float), hipMemcpyHostToDevice), "copy from data to gpu_data failed\n");
    //对于这个的coalesce不必要要求是128
    
    //printf("cpu calc#########################\n");
    //host_calcdtws(data,data,label,N,L,lo,hi,w,suma,sumb);
    

    int tempblocksize = 128;
    CheckCudaError(hipMalloc((void **)&gpu_label,tempblocksize*N*sizeof(char)),"cuda malloc gpu_label failed\n");
    CheckCudaError(hipMemcpy(gpu_label, label, tempblocksize*N*sizeof(char), hipMemcpyHostToDevice), "copy from label to gpu_label failed\n");
    
    float* gpu_infogain = NULL;
    float* gpu_dividepoint = NULL;
    int* gpu_leftis = NULL;
    int* gpu_glbidx = NULL;
    float* gpu_tempmin = NULL;
    int* gpu_tempminidx = NULL;

    CheckCudaError(hipMalloc((void **)&gpu_infogain,shapelet_num*sizeof(float)),"cuda malloc gpu_infogain failed\n");
    CheckCudaError(hipMalloc((void **)&gpu_dividepoint,shapelet_num*sizeof(float)),"cuda malloc gpu_dividepoint failed\n");
    CheckCudaError(hipMalloc((void **)&gpu_leftis,shapelet_num*sizeof(int)),"cuda malloc gpu_leftis failed\n");
    CheckCudaError(hipMalloc((void **)&gpu_glbidx,shapelet_num*sizeof(int)),"cuda malloc gpu_glbidx failed\n");
    CheckCudaError(hipMalloc((void **)&gpu_tempmin,shapelet_num*sizeof(float)),"cuda malloc gpu_tempmin failed\n");
    CheckCudaError(hipMalloc((void **)&gpu_tempminidx,shapelet_num*sizeof(int)),"cuda malloc gpu_tempminidx failed\n");


    float* gpu_cmpMinDist = NULL;
    int cmpMinDistSize = shapelet_num * coalesceforN * sizeof(float);
    CheckCudaError(hipMalloc((void **)&gpu_cmpMinDist, cmpMinDistSize),"cudamalloc gpu_cmpMinDist failed\n");

    float* gpu_fortranspose = NULL;
    int CoalescedForL_lo_1 = ((L-lo)/coalescefor128B + 1)*coalescefor128B;
    int fortransposeSize = idxstep * CoalescedForL_lo_1 * (hi-lo) * coalesceforN * sizeof(float);
    CheckCudaError(hipMalloc((void **)&gpu_fortranspose, fortransposeSize),"cudamalloc gpu_cmpMinDist failed\n");
    
    float lastinfogain = INFINITY;
  
    //STARTTIME;
    hiprandState_t* states;
    hipMalloc((void**) &states, shapelet_num * sizeof(hiprandState_t)); 
    rand_init<<<(shapelet_num-1)/256+1,256>>>(time(0),states, shapelet_num);
    int batch = 1;
   // int idxstep = shapelet_num/arrindex[L-lo+1];
    float elapsedtimefordis = 0.0;
    float elapsedtimeforinfo = 0.0;
    for(int idxst = 0; idxst < N; idxst += idxstep,batch++){
        int idxend = idxst + min(idxstep,N-idxst);
        int temp_shapelet_num = (idxend-idxst)*arrindex[L-lo+1];
        printf("*******The %d batch calc idx index from %d to %d, %d candidates in total.********\n",batch,idxst,idxend,temp_shapelet_num);
        Batch_CalcMinInfogain(gpu_data,gpu_label,gpu_cmpMinDist,gpu_infogain,gpu_dividepoint,gpu_leftis,gpu_glbidx,gpu_tempmin,gpu_tempminidx,w,N,L,lo,hi,temp_shapelet_num,outthrehold,&lastinfogain,p3,suma,sumb,states,idxst,idxend,func, gpu_arrindex, arrindex,&elapsedtimefordis,&elapsedtimeforinfo, gpu_fortranspose,CoalescedForL_lo_1);
    }
    
    //std::cout << "output:$$$$$$,w = " << w << std::endl;
    //std::cout << "output:$$$$$$,calc distance:" << elapsedtimefordis << ",calc infogain:" << elapsedtimeforinfo << std::endl;
    //STOPTIME;
    CheckCudaError(hipFree(gpu_cmpMinDist),"cudafree gpu_cmpMinDist failed\n");
    CheckCudaError(hipFree(gpu_fortranspose),"cudafree gpu_fortranspose failed\n");
    CheckCudaError(hipFree(gpu_data),"hipFree gpu_data failed\n");
    CheckCudaError(hipFree(gpu_label),"hipFree gpu_label failed\n");

    CheckCudaError(hipFree(gpu_infogain),"hipFree gpu_infogain failed\n");
    CheckCudaError(hipFree(gpu_dividepoint),"hipFree gpu_dividepoint failed\n");
    CheckCudaError(hipFree(gpu_leftis),"hipFree gpu_leftis failed\n");
    CheckCudaError(hipFree(gpu_glbidx),"hipFree gpu_glbidx failed\n");

    CheckCudaError(hipFree(gpu_tempmin),"hipFree gpu_tempmin failed\n");
    CheckCudaError(hipFree(gpu_tempminidx),"hipFree gpu_tempminidx failed\n");
    ENTIRE_STOPTIME;

    std::cout << "output:$$$$$$,w=" << w << ":" << ENTIRE_Mtime0 << "\t"<< elapsedtimefordis << "\t" << elapsedtimeforinfo << std::endl;
    return;

}
