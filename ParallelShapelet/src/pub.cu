#include "hip/hip_runtime.h"
// Copyright 2018 The Matrix Authors
// This file is part of the Matrix library.
//
// The Matrix library is free software: you can redistribute it and/or modify
// it under the terms of the GNU Lesser General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// The Matrix library is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU Lesser General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public License
// along with the Matrix library. If not, see <http://www.gnu.org/licenses/>.
/*
#include<stdio.h>
//#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""

#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/tuple.h>
*/
//#include <tuple>
#include ""
#include "pub.h"
#include <thrust/device_vector.h>
#include <thrust/pair.h>

#define M 100000000
//#define INFINITY 1.79769e+308

__host__ void CheckCudaError(hipError_t err, char const* errMsg){
    if(err == hipSuccess)
        return;
    printf("%s\nError Message:%s.\n",errMsg,hipGetErrorString(err));
    exit(EXIT_FAILURE);
}

__host__ __device__ int NumsOfThreadPerBlock(int needsize){
    int size = 32;
    while(size < needsize){
        size *= 2;
    }
    return size;
}

__host__ __device__ int index_compose(int n1,int D1,int n2, int D2 , int n3, int D3){
    return (n1*D2 + n2)*D3 + n3;
}

__host__ __device__ thrust::tuple<int,int,int> index_decompose(int index,int D1,int D2,int D3){
    int n3 = index % D3;
    int temp = index / D3;
    int n2 = temp % D2;
    int n1 = temp / D2;
    return thrust::make_tuple(n1,n2,n3);
}

__host__ __device__ int binarysearch(int* arr, int key, int low, int high){

    int mid;
    while(low <= high){
        mid = (low + high) >> 1;
        if(key == arr[mid])
            return mid;
        if(key < arr[mid]){
            high = mid - 1;
        }else{
            low = mid + 1;
        }
    }
    return low - 1;
}

__host__ __device__ int index_ComposeNoloss(int n1, int* arrindex, int IndexOflastElement, int n2, int n3){
    return n1 * arrindex[IndexOflastElement] + arrindex[n2] + n3;
}


__host__ __device__ thrust::tuple<int,int,int> index_decomposeNoloss(int index, int* arrindex, int IndexOflastElement){
    int n1 = index / arrindex[IndexOflastElement];
    int temp = index - n1 * arrindex[IndexOflastElement];
    int n2 = binarysearch(arrindex, temp , 0 , IndexOflastElement);
    int n3 = temp - arrindex[n2];
    return thrust::make_tuple(n1,n2,n3);
}


//N,0,0
/*
__host__ __device__ int index_CustomCompose(int idx,int st, int len, int N, int lo, int hi, int L){
    int 

}

__host__ __device__ thrust::tuple<int,int,int> index_Customdecompose(int index, int N, int L, int lo, int hi){


}
*/

/*********************************************************
public function calc the best infogain (include the below three function)
*********************************************************/
__host__ __device__ float Entropy(int  a, int b){
    if(a==0) return 0;
    if(b==0) return 0;
    float pa = (a+0.0)/(a+b+0.0);
    float pb = (b+0.0)/(a+b+0.0);
    return -(pa*log(pa)+pb*log(pb));
}


__host__ __device__ float EntropyScale(int a, int b){
    return (a+b)*Entropy(a,b);
}

//Entropy after division
__host__ __device__ float EntropySplit(int a1, int b1, int a2, int b2){
    return ((a1+b1)*Entropy(a1, b1)+(a2+b2)*Entropy(a2, b2))/(a1+a2+b1+b2);
}


__host__ __device__ float EntropyPN(int  a, int b, int p, int n){
    if(a==0) return 0;
    if(b==0) return 0;
    float pa = (a+0.0)/(a+b+0.0);
    float pb = (b+0.0)/(a+b+0.0);
    return -(p*pa*log(pa)+n*pb*log(pb));
}

//Entropy after division
__host__ __device__ float EntropySplitPN(int a1, int b1, int a2, int b2){
    return ((a1+b1)*Entropy(a1, b1)+(a2+b2)*Entropy(a2, b2))/(a1+a2+b1+b2);
}

//__host__ __device__ void swap(float* dist, int x, int y)

template <class T>
__host__ __device__ void swap(T& x,T& y)
{
     T temp;
     temp=x;
     x=y;
     y=temp;
}

__host__ __device__ int Partition(float* dist, char* label, int left, int right){
    
    float pivot = dist[right];
    int p = left;
    for(int q = left; q < right; q++){
        if(dist[q] < pivot){
            swap(dist[p],dist[q]);
            swap(label[p],label[q]);
            p ++;
        }
    }
    swap(dist[p],dist[right]);
    swap(label[p],label[right]);
    return p;
}


__host__ __device__ void quicksort(float* dist, char* label, int left, int right){

    if(left >= right)
        return;    
    int mid = Partition(dist,label,left,right);
    quicksort(dist,label,left,mid-1);
    quicksort(dist,label,mid+1,right);
}

/*
__host__ __device__ void quicksort(float* dist, char* label, thrust::device_vector<R>&Region){
//    thrust::device_vector<R> Region; 
    Region.push_back(thrust::make_pair(0,N-1));
    while(Region.size()>0){
        R it = Region.back();
        Region.pop_back();
        int left = it.first;
        int right = it.second;
        if(left >= right)
            continue;
        int mid = Partition(dist,label,left,right);
        Region.push_back(thrust::make_pair(left,mid-1));
        Region.push_back(thrust::make_pair(mid+1,right));
    }
}
*/
__host__ __device__ int RangeCount(char* label, int left, int right, char c){
    int cnt = 0;
    for(int i = left;i < right; i++)
        if(label[i] == c)
            cnt ++;
    return cnt;    
}


__host__ __device__ void HeuristicSplitInfogain(float* dist, int N, char* label, T3ddi* p1, int suma, int sumb){
    int lefta1,leftb1,righta1,rightb1;
    lefta1 = leftb1 = righta1 = rightb1 = 0;
    
    int left = 0; 
    int right = N - 1; 
    while(left < right){
        int p = Partition(dist,label,left,right);
        int lefta2 = RangeCount(label,left,p,0);
        int leftb2 = (p-left) - lefta2;
        float leftEntropy = Entropy(lefta1+lefta2,leftb1+leftb2);
        int righta2,rightb2;
        righta2 = suma - (lefta1+lefta2+righta1);
        rightb2 = sumb - (leftb1+leftb2+rightb1);
        if(label[p] == 0){
            righta2 -= 1;
        }
        else{
            rightb2 -= 1;
        }
        
        float rightEntropy = Entropy(righta1+righta2, rightb1+rightb2);
        if(leftEntropy < rightEntropy){
            lefta1 += lefta2;
            leftb1 += leftb2;
            if(label[p]==0){lefta1+=1;}else{leftb1+=1;}            
            left = p + 1;
        }else{
            righta1 += righta2;
            rightb1 += rightb2;
            if(label[p]==0){righta1+=1;}else{rightb1+=1;}
            right = p - 1;    
        }
    }
    //归入左边还是右边?
    int mida = (label[left]==0?1:0); 
    int midb = (label[left]==1?1:0);
    float goleft = EntropySplit(lefta1+mida,leftb1+midb,righta1,rightb1);
    float goright = EntropySplit(lefta1,leftb1,righta1+mida,rightb1+midb);
    float dividepoint;
    float infogain;
    int leftis;
    if(goleft < goright){
        float rightmin = INFINITY;
        for(int i = left + 1;i < N; i++){
            if(dist[i] < rightmin)
                rightmin = dist[i]; 
        }
        infogain = goleft;
        dividepoint = (rightmin+dist[left]) / 2;
        lefta1 += mida;
        leftb1 += midb;
        leftis = (lefta1>leftb1)?0:1;
    }else{
        float leftmax = 0;
        for(int i = left-1;i >= 0;i--){
            if(dist[i] > leftmax)
                leftmax = dist[i];
        }
        infogain = goright;
        dividepoint = (leftmax +dist[left]) / 2;
        righta1 += mida;
        rightb1 += midb;
        leftis = (lefta1>leftb1)?0:1;
    }   
    (*p1) = thrust::make_tuple(infogain,dividepoint,leftis);
}


//typedef thrust::tuple<float, float, int> T3ddi;
__host__ __device__ void BestSplitInfogain(float* dist, int N, char* label, T3ddi* p1, int suma, int sumb){

	int lefta1 = 0;
	int leftb1 = 0;
	
	float infogain = INFINITY;
	float dividepoint = 0.0;
	int leftis = 0;

	for(int i = 0;i< N-1;i++){
		if(label[i] == 0)
			lefta1 ++;
		else
			leftb1 ++;
		float thisentropy = EntropySplit(lefta1,leftb1,suma-lefta1,sumb-leftb1);
//		float thisgap = dist[i+1] - dist[i];
//        thisentropy = thisentropy * M + ((thisgap>10)?0:(10-thisgap)); 
		if(thisentropy < infogain){
			infogain = thisentropy;
			leftis = (lefta1>leftb1)?0:1;
			dividepoint = (dist[i]+dist[i+1])/2;
		}
	}
    (*p1) = thrust::make_tuple(infogain,dividepoint,leftis);
}

__host__ __device__ void BestSplitInfogainNoStrust(float* dist, int N, int* label, float* infogain_out, float* dividepoint_out, int*leftis_out){

	int suma = 0;
	int sumb = 0;
	for(int i = 0;i < N;i++){
		if(label[i] == 0)
			suma ++;
		else
			sumb ++;
	}
	int lefta1 = 0;
	int leftb1 = 0;
	
	float infogain = INFINITY;
	float dividepoint = 0.0;
	int leftis = 0;

	for(int i = 0;i< N-1;i++){
		if(label[i] == 0)
			lefta1 ++;
		else
			leftb1 ++; 
		float thisentropy = EntropySplit(lefta1,leftb1,suma-lefta1,sumb-leftb1);
		
		if(thisentropy < infogain){
			infogain = thisentropy;
			leftis = (lefta1>leftb1)?0:1;
			dividepoint = (dist[i]+dist[i+1])/2;
		}
	}
    //(*p1) = thrust::make_tuple(infogain,dividepoint,leftis);
    (*infogain_out) = infogain;
    (*dividepoint_out) = dividepoint;
    (*leftis_out) = leftis;
}




