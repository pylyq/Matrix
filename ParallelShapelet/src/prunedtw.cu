#include "hip/hip_runtime.h"
// Copyright 2018 The Matrix Authors
// This file is part of the Matrix library.
//
// The Matrix library is free software: you can redistribute it and/or modify
// it under the terms of the GNU Lesser General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// The Matrix library is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU Lesser General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public License
// along with the Matrix library. If not, see <http://www.gnu.org/licenses/>.
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include "pub.h"
#include "prunedtw.h"

#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/tuple.h>
#include <thrust/fill.h>
//#define INFINITY 1.79769e+308
#define abs(x) (x>0)?x:-x

#define square(x) ((x)*(x))

#define BankConflictStride 17
//#define Coalescedfor128B 128/sizeof(float)
const int Coalescedfor128B = 128/sizeof(float);

typedef thrust::tuple<float,float,int> T3ddi;


/********************************************
Prune implemented dtw on cpu,but length
*********************************************/
//参数部分可以放在constant device中
__device__ bool isvalid(int x,int y, int w, int M){
    return (y>=max(1,x-w) && (y<=min(M,x+w))&& (x>=1) && (x<=M));
}
/***********************************************
when transformation between no diag element and diag element;
***********************************************/
#define Reverse  if(turn == 0){                 \
                        x0 += 1;                \
                        turn = 1;               \
                    }                           \
                    else{                       \
                        y0 += 1;                \
                        turn = 0;               \
                    }                           \
                    firstline = secondline;     \
                    secondline = thirdline;     \
                    thirdline = firstline;
/*************************************************
when w < threadhold and N < threadhold;
*************************************************/
__global__ void kernel_Computedtwsperblock(float* data, float* shapelet, int w, int N, int L, int lo, int hi, float* gpu_cmpMinDist, int idxst, int* gpu_arrindex){
    //这个函数的参数要好好设计一下
    extern __shared__ float cache[];

    int grpidx = blockIdx.x + idxst;
    int grpstx = blockIdx.y;

    int idx = threadIdx.x;
    int tid = threadIdx.y;
    
    //coalesceforN 由blockDim.x传递
    int coalesceforN = ((N-1)/Coalescedfor128B+1)*Coalescedfor128B;
    //griddim.y = L-lo +1
    //float* glb2local = gpu_cmpMinDist + ((grpidx-idxst) * gridDim.y + grpstx) * (hi-lo) * coalesceforN;
    int* cache_arrindex = (int*)(cache + 2*N*(w+2));
    int step1 = blockDim.x * blockDim.y;

    for(int i = threadIdx.y * blockDim.x + threadIdx.x; i < L; i+= step1){
        cache_arrindex[i] = gpu_arrindex[i];
    }
    
    __syncthreads();
    //float* glb2local = gpu_cmpMinDist + ((grpidx-idxst) * cache_arrindex[L-lo+1] + cache_arrindex[grpstx]) * coalesceforN;
    float* glb2local = gpu_cmpMinDist + index_ComposeNoloss(grpidx-idxst, cache_arrindex, L-lo+1, grpstx, 0) * coalesceforN;
    int end = (cache_arrindex[grpstx+1] - cache_arrindex[grpstx]) * coalesceforN;
    for(int i = threadIdx.y * blockDim.x + threadIdx.x; i < end; i+= step1){
        glb2local[i] = INFINITY;
    }    
    __syncthreads();
    
    
    if(idx < N && grpstx < (L-lo+1)){


        int wide = w + 1;
        int size = (w + 2)*2;
        int center = wide/2;

        float* sdata = cache + idx * size;

        float* tmpshapelet = shapelet + grpidx * L + grpstx;
        
        //public
        //虽然 griddim.y = L - lo + 1,但是gridDim.y对应grpstx,不应该在这里使用.
        for(int st = 0; st < L - lo + 1; st++){

            //data[idx][st:(st+len)]            
            float* tmp_data = data + (idx*L) + st;
            {
                for(int i = tid;i < size;i+= blockDim.y)
                    sdata[i] = INFINITY;
                if(tid == 0)
                    sdata[center] = 0.0;
            }
            __syncthreads();

            
            int x0,y0,x,y,turn;
            x0 = 1 - (w >> 1); y0 = 1 + (w >> 1);
            turn = w&(0x01);

            float* firstline = sdata;
            float* secondline = sdata + wide + 1;
            float* thirdline = firstline;
            
            //modify @1.22 add L-st+1,modify @ 1.17;
            //DTW截至长度.
            int longest = min(hi, min(L-grpstx+1, L-st+1));
            for(int step = 1; step < longest; step++){
                
                __syncthreads();
                for(int i = tid;i < wide; i+= blockDim.y){
                    x = x0 + i;
                    y = y0 - i;
                    
                    if(isvalid(x,y,w,longest)){
                        // modify @ 1.22
                        thirdline[i+turn] = square(tmp_data[x-1]-tmpshapelet[y-1]) + min(firstline[i+turn],min(secondline[i],secondline[i+1]));
                    }
                }

                //modify @ 1.22 avoid the 1-31 threads arrive early.
                __syncthreads();
                
                //这里限制了带宽,使带宽降低了16倍
                //Bandrate 85GB/s
                if((tid==0)&&(step>=lo)){
                    int tmp_index = (step-lo) * coalesceforN + idx;
                    glb2local[tmp_index] = min(sdata[center], glb2local[tmp_index]);
                    if(grpidx == 22 && grpstx == 1 && step == 42 && st == L - lo){
                        printf("gpu:%f\n",glb2local[tmp_index]);
                    }
                }
                // avoid the last threads arrive early. @1.22
                __syncthreads();
                Reverse;

                __syncthreads();
                for(int i = tid;i < wide; i+= blockDim.y){
                    x = x0 + i;
                    y = y0 - i;
                    if(isvalid(x,y,w,longest)){
                        thirdline[i+turn] = square(tmp_data[x-1]-tmpshapelet[y-1]) + min(firstline[i+turn],min(secondline[i],secondline[i+1]));
                    }
               
                }
                __syncthreads();
                Reverse;
            }
            
        }

    }
}

/***********************************************
  for big data. 
***********************************************/
__global__ void kernel_ComputedtwsperblockforlargerW(float* data, float* shapelet, int w, int N, int L, int lo, int hi, float* gpu_cmpMinDist, int idxst, int* gpu_arrindex){
    //这个函数的参数要好好设计一下
    extern __shared__ float cache[];

    int grpidx = blockIdx.x + idxst;
    int grpstx = blockIdx.y;

    int idx = threadIdx.x;
    int tid = threadIdx.y;
    int* cache_arrindex = (int*)(cache + 2*(w+2)*blockDim.x);
    int coalesceforN = ((N - 1)/Coalescedfor128B + 1) * Coalescedfor128B;
    int step1 = blockDim.x * blockDim.y;
    for(int i = threadIdx.y * blockDim.x + threadIdx.x; i < L; i+= step1){
        cache_arrindex[i] = gpu_arrindex[i];        
    }
    __syncthreads();
    //float* glb2local = gpu_cmpMinDist + (grpidx * (L - lo +  1) + grpstx) * (hi-lo) * coalesceforN;
    //float* glb2local = gpu_cmpMinDist + ((grpidx - idxst) * gridDim.y + grpstx) * (hi-lo) * coalesceforN;
    //float* glb2local = gpu_cmpMinDist + ((grpidx - idxst) * cache_arrindex[L-lo+1] + cache_arrindex[grpstx])  * coalesceforN;
    float* glb2local = gpu_cmpMinDist + index_ComposeNoloss(grpidx-idxst, cache_arrindex, L-lo+1, grpstx, 0) * coalesceforN;
    //int end = (hi - lo) * coalesceforN;
    int end = (cache_arrindex[grpstx+1] - cache_arrindex[grpstx]) * coalesceforN;
    for(int i = threadIdx.y * blockDim.x + threadIdx.x; i < end; i+= step1){
        glb2local[i] = INFINITY;
    }    
    __syncthreads();
    
    int wide = w + 1;
    int size = (w + 2)*2;
    int center = wide/2;

    float* sdata = cache + idx * size;
//  float* transfer = cache + N*2*(w+2);
//  add @ 1.17,update @ 1.20
   
    //modify for big data @1.25 remove idx < N
    //其实可以同一个代码?对吧? 
    if(grpstx < (L-lo+1)){

        float* tmpshapelet = shapelet + grpidx * L + grpstx;
        
        for(; idx < N; idx += blockDim.x){
        //public
            for(int st = 0; st < L - lo + 1; st++){
                
                float* tmp_data = data + (idx*L) + st;
                {
                    for(int i = tid;i < size;i+= blockDim.y)
                        sdata[i] = INFINITY;
                    if(tid == 0)
                        sdata[center] = 0.0;
                }

                __syncthreads();
                
                int x0,y0,x,y,turn;
                x0 = 1 - (w / 2); y0 = 1 + (w / 2);
                turn = w&(0x01);

                float* firstline = sdata;
                float* secondline = sdata + wide + 1;
                float* thirdline = firstline;
                
                //modify @1.22 add L-st+1,modify @ 1.17;
                int longest = min(hi, min(L-grpstx+1, L-st+1));
                for(int step = 1; step < longest; step++){
                    
                    __syncthreads();
                    for(int i = tid;i < wide; i+= blockDim.y){
                        x = x0 + i;
                        y = y0 - i;
                        if(isvalid(x,y,w,hi)){
                            // modify @ 1.22
                            thirdline[i+turn] = square(tmp_data[x-1]-tmpshapelet[y-1]) + min(firstline[i+turn],min(secondline[i],secondline[i+1]));

                        }
                    }

                    //modify @ 1.22 avoid the 1-31 threads arrive early.
                    __syncthreads();
                    
                    //这里限制了带宽,使带宽降低了16倍
                    if((tid==0)&&(step>=lo)){
                        int tmp_index = (step-lo) * coalesceforN + idx;
                        glb2local[tmp_index] = min(sdata[center], glb2local[tmp_index]);
                        /*
                        if(grpidx == 3 && grpstx == 27 && idx == 3 && step == 86){
                            printf("gpu list :st = %d,%lf,%lf\n",st,sdata[center],glb2local[tmp_index]);
                        }
                        */
                    }
     
                    // avoid the last threads arrive early. @1.22
                    __syncthreads();
                    Reverse;

                    __syncthreads();
                    for(int i = tid;i < wide; i+= blockDim.y){
                        x = x0 + i;
                        y = y0 - i;
                        if(isvalid(x,y,w,hi)){
                            thirdline[i+turn] = square(tmp_data[x-1]-tmpshapelet[y-1]) + min(firstline[i+turn],min(secondline[i],secondline[i+1]));
                        }
                   
                    }
                    __syncthreads();
                    Reverse;
                }
                
            }
            /*
            if(grpidx == 3 && grpstx == 27 && idx == 3 && tid == 0){
                printf("the 3 distance is %lf\n",gpu_cmpMinDist[21646*64+idx]);
            }
            */
        }
    }
}


__global__ void kernel_ComputedtwsperblockforAll(float* data, float* shapelet, int w, int N, int L, int lo, int hi, float* gpu_cmpMinDist, int idxst, int* gpu_arrindex){
    //这个函数的参数要好好设计一下
    extern __shared__ float cache[];

    int grpidx = blockIdx.x + idxst;
    int grpstx = blockIdx.y;

    int idx = threadIdx.x;
    int tid = threadIdx.y;

    int stride = 2*(w+2)+1;
    int* cache_arrindex = (int*)(cache + stride *blockDim.x);
    //int coalesceforN = (N - 1)&0xFFF0 + 0x0010;
    int coalesceforN = ((N - 1)/Coalescedfor128B + 1) * Coalescedfor128B;

    int step_CoalescedCopy = blockDim.x * blockDim.y;
    for(int i = threadIdx.y * blockDim.x + threadIdx.x; i < L; i+= step_CoalescedCopy){
        cache_arrindex[i] = gpu_arrindex[i];        
    }
    __syncthreads();
    //float* glb2local = gpu_cmpMinDist + (grpidx * (L - lo +  1) + grpstx) * (hi-lo) * coalesceforN;
    //float* glb2local = gpu_cmpMinDist + ((grpidx - idxst) * gridDim.y + grpstx) * (hi-lo) * coalesceforN;
    //float* glb2local = gpu_cmpMinDist + ((grpidx - idxst) * cache_arrindex[L-lo+1] + cache_arrindex[grpstx])  * coalesceforN;
    float* glb2local = gpu_cmpMinDist + index_ComposeNoloss(grpidx-idxst, cache_arrindex, L-lo+1, grpstx, 0) * coalesceforN;
    //int end = (hi - lo) * coalesceforN;
    int end = (cache_arrindex[grpstx+1] - cache_arrindex[grpstx]) * coalesceforN;
    for(int i = threadIdx.y * blockDim.x + threadIdx.x; i < end; i+= step_CoalescedCopy){
        glb2local[i] = INFINITY;
    }    
    __syncthreads();
    
    int wide = w + 1;
    int size = (w + 2)*2;
    int center = wide/2;

    float* sdata = cache + idx * stride;
    
    if(grpstx < (L-lo+1)){

        float* tmpshapelet = shapelet + grpidx * L + grpstx;
        //When N < blockDim.x, Only 1 loop for every thread.
        for(; idx < N; idx += blockDim.x){
        //if(idx < N){
            for(int st = 0; st < L - lo + 1; st++){
                
                float* tmp_data = data + (idx*L) + st;
                {
                    for(int i = tid;i < size;i+= blockDim.y)
                        sdata[i] = INFINITY;
                    if(tid == 0)
                        sdata[center] = 0.0;
                }

                __syncthreads();
                
                int x0,y0,x,y,turn;
                x0 = 1 - (w >> 1); y0 = 1 + (w >> 1);
                turn = w&(0x01);

                float* firstline = sdata;
                float* secondline = sdata + wide + 1;
                float* thirdline = firstline;
                
                //modify @1.22 add L-st+1,modify @ 1.17;
                int longest = min(hi, min(L-grpstx+1, L-st+1));
                for(int step = 1; step < longest; step++){
                    
                    __syncthreads();
                    for(int i = tid;i < wide; i+= blockDim.y){
                        x = x0 + i;
                        y = y0 - i;
                        if(isvalid(x,y,w,longest)){
                            // modify @ 1.22
                            thirdline[i+turn] = square(tmp_data[x-1]-tmpshapelet[y-1]) + min(firstline[i+turn],min(secondline[i],secondline[i+1]));

                        }
                    }
                    //modify @ 1.22 avoid the 1-31 threads arrive early.
                    __syncthreads();
                    
                    //这里限制了带宽,使带宽降低了16倍
                    if((tid==0)&&(step>=lo)){
                        int tmp_index = (step-lo) * coalesceforN + idx; 
                        glb2local[tmp_index] = min(sdata[center], glb2local[tmp_index]);
                        /*
                        if(grpidx == 22 && grpstx == 0 && step == 42 && idx == 1 && tid == 0){
                            printf("gpu:%f\n",glb2local[tmp_index]);
                        }
                        */
                    }
     
                    // avoid the last threads arrive early. @1.22
                    __syncthreads();
                    Reverse;

                    __syncthreads();
                    for(int i = tid;i < wide; i+= blockDim.y){
                        x = x0 + i;
                        y = y0 - i;
                        if(isvalid(x,y,w,longest)){
                            thirdline[i+turn] = square(tmp_data[x-1]-tmpshapelet[y-1]) + min(firstline[i+turn],min(secondline[i],secondline[i+1]));
                        }
                   
                    }
                    __syncthreads();
                    Reverse;
                }
                
            }
            /*
            if(grpidx == 22 && grpstx == 1 && idx == 0 && tid == 0){
            float* ccccc = gpu_cmpMinDist + index_ComposeNoloss(grpidx-idxst, cache_arrindex, L-lo+1, grpstx, (42-lo)) * coalesceforN;
            for(int i = 0;i < N;i++)
                printf("gpu:%f\n",ccccc[i]);
            }
            */
        }


        
    }
}
