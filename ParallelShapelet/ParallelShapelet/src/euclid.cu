#include "hip/hip_runtime.h"
// Copyright 2018 The Matrix Authors
// This file is part of the Matrix library.
//
// The Matrix library is free software: you can redistribute it and/or modify
// it under the terms of the GNU Lesser General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// The Matrix library is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU Lesser General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public License
// along with the Matrix library. If not, see <http://www.gnu.org/licenses/>.
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include<stdio.h>
//#include <stdlib.h> 
#include <time.h>  
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include "euclid.h"

const int coalescefor128B = 128/sizeof(float);
//const unsigned int Mask_0xFFF0 = 0xFFF0;


__global__ void fillarrwith(float* gpu_arr, int D1, float value){

    int tid = threadIdx.x;
    int index_out = blockIdx.x * blockDim.x + tid;
    if(index_out < D1){
        gpu_arr[index_out] = value;
    }
}

__global__ void kernel_ComputeDist(float* gpu_data, float* gpu_infogain, float* gpu_dividepoint,int* gpu_leftis,const int N,const int L, const int lo, const int hi, const float outthrehold, float* gpu_cmpMinDist, int idxst,  int* gpu_arrindex){

    int s = threadIdx.x; 
    int idx = blockIdx.x + idxst;
    int len = blockIdx.y + lo;
    int stop = L-len+1;

    extern __shared__ char smem[];

    float* buf0 = (float*)smem;
    float* buf = buf0 + L;
    int* cache_arrindex = (int*)(smem + 2 * L * sizeof(float));
    
    for(int i = s; i < L; i += blockDim.x){
        cache_arrindex[i] = gpu_arrindex[i];
    }
    
    __syncthreads();

    if(idx < N && len < hi && s < stop){
     
        int coalesceforN = ((N-1)/coalescefor128B+1)*coalescefor128B;
        //int coalesceforN = ((N-1)/coalescefor128B)+coalescefor128B;

//      int temp = ((idx * (L-lo+1) + s)*(hi-lo)+(len-lo));
//      int index_out = index_compose(idx - idxst,N,s,L-lo+1,len-lo,hi-lo);
//        int index_out = (idx - idxst)*gpu_arrindex[L-lo+1]+gpu_arrindex[s]+(len-hi);
//        int index_out = (idx - idxst)*cache_arrindex[L-lo+1] + cache_arrindex[s]+(len-lo);
        int index_out = index_ComposeNoloss(idx-idxst, cache_arrindex, L-lo+1, s, len-lo);
//        float* glb2local = gpu_cmpMinDist + ((idx * (L-lo+1) + s)*(hi-lo)+(len-lo))*coalesceforN;
        float* glb2local = gpu_cmpMinDist + index_out*coalesceforN;

        float dist[coalescefor128B];
        //last:float dist[77];
        float tmp;
        float tmp_l;
        int coal_j = 0;
        for(int j = 0 ;j < N; j++){
            
//           label[j] = gpu_label[j];
            buf0[s] = 0.0;
            buf[s] = 0.0;
            
            __syncthreads();
            int idxXL = idx * L;
            int jXL = j * L;
            for(int i = 0; i < len; i++){
                //Omega,(idx,1),(j,s+1)
                tmp = (gpu_data[idxXL+i] - gpu_data[jXL+s+i]);               
                buf0[s] += tmp * tmp;
                //Omega,(idx,s+1),(j,1)
                tmp = gpu_data[idxXL+s+i] - gpu_data[jXL+i];
                buf[s] += tmp * tmp;
            }
            
            dist[coal_j] = buf[s];
            
            //last:dist[j] = buf[s];
            
            for(int u = 1;u < L-len+1;u ++){
                //O(1)
                __syncthreads();
                
                if(s!=0){
                    tmp = gpu_data[idxXL+s-1] - gpu_data[jXL+u-1];
                    tmp_l = gpu_data[idxXL+s-1+len] - gpu_data[jXL+u-1+len];
                    buf[s] = buf[s-1] + (tmp_l* tmp_l) - (tmp * tmp);
                }
                __syncthreads();

               if(s==0)
                    buf[s] = buf0[u];

                if(buf[s] < dist[coal_j])
                    dist[coal_j] = buf[s];
            }

            coal_j ++;
            if(coal_j == coalescefor128B || j == N-1){
                for(int u=0;u < coalescefor128B;u++){
                    glb2local[u] = dist[u];
                }
                glb2local += coalescefor128B;
                coal_j = 0; 
            }

        }
    }
}


__global__ void kernel_ComputeDistSaveBlk(float* gpu_data, const int N,const int L, const int lo, const int hi,  float* gpu_fortranspose, int idxst,  int* gpu_arrindex, int CoalescedForL_lo_1){

    int s = threadIdx.x; 
    int idx = blockIdx.x + idxst;
    int len = blockIdx.y + lo;
    int stop = L-len+1;

    extern __shared__ char smem[];

    float* buf0 = (float*)smem;
    float* buf = buf0 + L;
    float* maxbuf = buf + L;

    int coalesceforN = ((N-1)/coalescefor128B+1)*coalescefor128B;
    float* gpu_transform = gpu_fortranspose + (blockIdx.x + blockIdx.y * gridDim.x)*CoalescedForL_lo_1*coalesceforN;

    if(idx < N && len < hi && s < stop){
     
        //float* glb2local = gpu_cmpMinDist + index_out*coalesceforN;

        //float dist[coalescefor128B];
        //last:float dist[77];
        float tmp;
        float tmp_l;
        
        for(int j = 0 ;j < N; j++){
            
            buf0[s] = 0.0;
            buf[s] = 0.0;
            
            __syncthreads();
            int idxXL = idx * L;
            int jXL = j * L;
            for(int i = 0; i < len; i++){
                //Omega,(idx,1),(j,s+1)
                tmp = (gpu_data[idxXL+i] - gpu_data[jXL+s+i]);               
                buf0[s] += tmp * tmp;
                //Omega,(idx,s+1),(j,1)
                tmp = gpu_data[idxXL+s+i] - gpu_data[jXL+i];
                buf[s] += tmp * tmp;
            }
            
            maxbuf[s] = buf[s];
            
            for(int u = 1;u < L-len+1;u ++){
                //O(1)
                __syncthreads();
                
                if(s!=0){
                    tmp = gpu_data[idxXL+s-1] - gpu_data[jXL+u-1];
                    tmp_l = gpu_data[idxXL+s-1+len] - gpu_data[jXL+u-1+len];
                    buf[s] = buf[s-1] + (tmp_l* tmp_l) - (tmp * tmp);
                }
                __syncthreads();

               if(s==0)
                    buf[s] = buf0[u];

                if(buf[s] < maxbuf[s])
                    maxbuf[s] = buf[s];
            }
            
            gpu_transform[j* CoalescedForL_lo_1 + s] = maxbuf[s];
        }
    }
}

__global__ void kernel_Transpose(float* gpu_fortranspose, float* gpu_cmpMinDist, const int N, const int L, const int lo, const int hi, int idxst, int* gpu_arrindex, int coalesceforN, int CoalescedForL_lo_1){

    int s = threadIdx.x + blockDim.x * threadIdx.y; 
    int idx = blockIdx.x + idxst;
    int len = blockIdx.y + lo;
    int stop = L - len + 1;

    //Coalesced XiuGai
    float* gpu_transform = gpu_fortranspose + (blockIdx.x + blockIdx.y * gridDim.x)* CoalescedForL_lo_1 *coalesceforN;
    //L-len+1
    //33*(L-lo+1);
    extern __shared__ char smem[];

    int* cache_arrindex = (int*)(smem);
    
    for(int i = s; i < L; i += blockDim.x* blockDim.y){
        cache_arrindex[i] = gpu_arrindex[i];
    }

    float* sdata = ((float*)smem) + L;

    //Coalesced XiuGai
    //float* gpu_temp = gpu_transform;// (L-len+1)*coalescefor128B;
//    int stop = L-len+1;
    for(int offset = 0; offset < coalesceforN; offset += coalescefor128B){
    //    float* gpu_temp = gpu_transform + (L-len+1)*coalescefor128B;
        for(int i = 0; i < coalescefor128B; i+= 1){
            if(s < stop){
                sdata[s*33+i] = gpu_transform[(offset+i)* CoalescedForL_lo_1 + s];
            }
            __syncthreads();
        }
        for(int row = 0;row < stop; row += blockDim.y){
            int index_out = index_ComposeNoloss(idx-idxst, cache_arrindex, L-lo+1, row, len-lo);
            float* glb2local = gpu_cmpMinDist + index_out * coalesceforN + offset;
            glb2local[threadIdx.x] = sdata[row*33+threadIdx.x];
        }
    }
}
