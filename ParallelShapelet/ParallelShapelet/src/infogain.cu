#include "hip/hip_runtime.h"
// Copyright 2018 The Matrix Authors
// This file is part of the Matrix library.
//
// The Matrix library is free software: you can redistribute it and/or modify
// it under the terms of the GNU Lesser General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// The Matrix library is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU Lesser General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public License
// along with the Matrix library. If not, see <http://www.gnu.org/licenses/>.
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include "pub.h"
#include "prunedtw.h"

#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/tuple.h>
#include <thrust/fill.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/pair.h>


#define INFOGAINBLOCKSIZE 32


__global__ void rand_init(unsigned int seed, hiprandState_t* states, int shapelet_num){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < shapelet_num){
        hiprand_init(seed, blockIdx.x, 0, &states[blockIdx.x]);
    }
}

template <class T>
__host__ __device__ void swap(T& x,T& y)
{
    T temp;
    temp=x;
    x=y;
    y=temp;
}

__global__ void kernel_Computeifgain(float* gpu_cmpMinDist, char* gpu_label, const int N, const int totals, float* gpu_infogain, float* gpu_dividepoint, int* gpu_leftis, int* gpu_glbidx, const float outthrehold, const int L, const int lo, const int hi, const int labelcacheoffset, const int coalesceforN, int suma, int sumb, hiprandState_t* states, int w, int idxst, int* gpu_arrindex){
    //coalesceforN read a group of distance. 
    
    int tid = threadIdx.x;
    int s = threadIdx.y;
    int index = blockIdx.x * blockDim.y + s;
    float* gpu_local = gpu_cmpMinDist + index * coalesceforN;
        
    extern __shared__ char cache[];

    int end = INFOGAINBLOCKSIZE * N;
    int step = blockDim.x * blockDim.y;
    for(int i = tid + s * blockDim.x; i < end; i += step){
        cache[i] = gpu_label[i];
    }
    
    int* cacheindex = (int*)(cache+end);
//    __syncthreads();
    for(int i = tid + s * blockDim.x; i < L; i+= step){
        cacheindex[i] = gpu_arrindex[i];    
    }
    
    __syncthreads();
    
    if(tid ==0 && index < totals){
            
        char* locallabel = cache + s * N;
        T3ddi p1;

        /*
        for(unsigned int i = 1;i < N;i++){
            unsigned int j = hiprand(&states[index])%(i+1);
            if(i==j)
                continue; 
            swap(gpu_local[i], gpu_local[j]);
            swap(locallabel[i], locallabel[j]);
        }
        HeuristicSplitInfogain(gpu_local, N, locallabel, &p1, suma, sumb);
        */
        
        thrust::sort_by_key(thrust::seq, gpu_local, gpu_local + N, locallabel);
        BestSplitInfogain(gpu_local, N, locallabel, &p1,suma,sumb);
        
        //    thrust::tuple<int,int,int> n123 = index_decompose(index,N,hi-lo,L-lo+1);
        //    idx = thrust::get<0>(n123);
        //    st = thrust::get<2>(n123);
        //    len = thrust::get<1>(n123) + lo;

        gpu_infogain[index] = thrust::get<0>(p1);
        gpu_dividepoint[index] = thrust::get<1>(p1);
        gpu_leftis[index] = thrust::get<2>(p1);

        if(thrust::get<0>(p1) < outthrehold){
             //使用incompose做一下.
             thrust::tuple<int,int,int> n123 = index_decomposeNoloss(index, cacheindex, L-lo+1);
             int idx = thrust::get<0>(n123) + idxst;
             int st = thrust::get<1>(n123);
             int len = thrust::get<2>(n123) + lo;
        /*
             int idx = index/cacheindex[L-lo+1] + idxst;
             int temp = index - (idx-idxst)*cacheindex[L-lo+1];
             int st = binarysearch(cacheindex,temp,0,L-lo+1);
         */
             /*
             int st = 0;
             while(cacheindex[st+1] <= temp){
                st ++;
             }
             */
//             int len = temp - cacheindex[st] + lo;
             
             printf("infogain = %lf,dividepoint = %lf, leftis = %d, idx = %d, st = %d, len = %d, index_out = %d,(%d,%d,%d,%d)\n",thrust::get<0>(p1) , thrust::get<1>(p1),thrust::get<2>(p1),idx,st,len,index,lo,hi,N,L);
        }
    
    }

}

