#include "hip/hip_runtime.h"
// Copyright 2018 The Matrix Authors
// This file is part of the Matrix library.
//
// The Matrix library is free software: you can redistribute it and/or modify
// it under the terms of the GNU Lesser General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// The Matrix library is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU Lesser General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public License
// along with the Matrix library. If not, see <http://www.gnu.org/licenses/>.
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include "pub.h"
#include "distanceoncpu.h"

#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/tuple.h>
#include <thrust/fill.h>
//#define INFINITY 1.79769e+308
using namespace std;

#define abs(x) (x>0)?x:-x

#define square(x) ((x)*(x))

typedef thrust::tuple<float,float,int> T3ddi;
// function declaration
//__host__ __device__ void BestSplitInfogain(float* dist, int N, int* label, T3ddi* p1);
/********************************************
native implemented dtw on cpu
*********************************************/

__host__ float native_dtw(float* vec_a,float* vec_b, int M){

    float dp[M+1][M+1];
	for(int i = 1;i <= M;i++){
        dp[i][0] = INFINITY;
        dp[0][i] = INFINITY;
    }

    dp[0][0] = 0.0;
    for(int m = 1;m <= M;m++){
        for(int n = 1;n <= M;n++){
            dp[m][n] = square(vec_a[m-1]-vec_b[n-1]) + min(dp[m-1][n-1],min(dp[m-1][n],dp[m][n-1]));
//            printf(" %lf",dp[m][n]);
        }
//        printf("\n");
    }
    return dp[M][M];
}


/********************************************
pruned implemented dtw on cpu
*********************************************/

bool isvalidoncpu(int x,int y, int w, int M){
    return (y>=max(1,x-w) && (y<=min(M,x+w))&& (x>=1) && (x<=M));
}
__host__ float Euclid_Distance(float* vec_a, float* vec_b, int M){
    float tmp = 0.0;
    for(int i = 0; i < M; i++){
        tmp += square(vec_a[i]-vec_b[i]);
    }
    return tmp;
}


__host__ float Pruning_dtw(float* vec_a, float* vec_b, int w, int M){

	float dp[M+1][M+1];
    //avoid some -nan;
    memset(dp,0,(M+1)*(M+1)*sizeof(float));

    dp[0][0] = 0.0;
	for(int i = 1;i <= M;i++){
        dp[i][0] = INFINITY;
        dp[0][i] = INFINITY;
    }
    
    float left,down;
    for (int i = 1; i <= M; i++)
    {
        for (int j = max(1, i - w); j <= min(M, i + w); j++)
        {
            left = isvalidoncpu(i,j-1,w,M)?dp[i][j-1]:INFINITY;
            down = isvalidoncpu(i-1,j,w,M)?dp[i-1][j]:INFINITY;
            dp[i][j] = min(dp[i-1][j-1],min(left,down)) + square(vec_a[i-1]-vec_b[j-1]);
        }
    }
    return dp[M][M];
}

typedef thrust::tuple<int,int,int> T3iloc;

__host__ void host_calcdtws(float* data, float* shapelet, char* label,int N, int L, int lo, int hi, int w, int suma, int sumb, TPOUT* p3){
   
    T3iloc lastloc = thrust::make_tuple(0,0,0);
    T3ddi  lastifgain = thrust::make_tuple(INFINITY,INFINITY,0);
    T3ddi  tempifgain = thrust::make_tuple(INFINITY,INFINITY,0);
    for(int grpidx = 0; grpidx < N; grpidx++){
        for(int grpstx = 0; grpstx < L-lo+1; grpstx++){
            for(int len = lo; len < min(hi,L-grpstx+1); len ++){

                if(grpidx != 22 || grpstx != 1 || len != 42)
                   continue;
                //printf("hello world.\n");
                float* tmpshapelet = shapelet + grpidx*L + grpstx;
                float dist[N];
                char tmplabel[N];
                copy(label, label+N, tmplabel);
                for(int idx = 0; idx < N; idx++){
                    dist[idx] = INFINITY;
                    
                    for(int i = 0; i < L-len+1;i++){
                        float* tmpdata = data + idx* L + i;
                        float tmpdis;
                        if(w == 0){
                            tmpdis = Euclid_Distance(tmpdata,tmpshapelet,len);
                        }
                        else{
                            tmpdis = Pruning_dtw(tmpdata,tmpshapelet,w,len);
                        }
                        dist[idx] = min(dist[idx],tmpdis);
                    }
                    printf("cpu:%f,%d\n",dist[idx],label[idx]);
                }
                //排序.
                thrust::stable_sort_by_key(thrust::host, dist, dist+N, tmplabel);
                BestSplitInfogain(dist,N,tmplabel,&tempifgain,suma,sumb);
//                HeuristicSplitInfogain(dist,N,label,&tempifgain,suma,sumb);
                /* 
                for(int i = 0;i < N;i++){
                    printf("cpu:%lf\n",dist[i]);
                }
               */ 
                //if(grpidx == 13 && grpstx == 1 && len == 40)
                //    printf("13140:%lf\n",thrust::get<0>(tempifgain));
                if(thrust::get<0>(tempifgain) < thrust::get<0>(lastifgain)){
                    lastifgain = tempifgain;
                    lastloc = thrust::make_tuple(grpidx,grpstx,len);
                }
                //printf("######################################the cpu calc ifgain = %lf, dividepoint = %lf\n",thrust::get<0>(lastifgain),thrust::get<1>(lastifgain));
            }    
        }
    }
    (*p3) = thrust::make_tuple(thrust::get<0>(lastloc),thrust::get<2>(lastloc),thrust::get<1>(lastloc),thrust::get<0>(lastifgain),thrust::get<1>(lastifgain),thrust::get<2>(lastifgain));

    printf("The CPU calc infogain = %lf,dividepoint = %lf,location = (%d,%d,%d)\n",thrust::get<0>(lastifgain),thrust::get<1>(lastifgain),thrust::get<0>(lastloc),thrust::get<1>(lastloc),thrust::get<2>(lastloc));
}

